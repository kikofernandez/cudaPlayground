// Square Matrix addition


#include <hip/hip_runtime.h>
#include <iostream>

#define N (20)
#define NBlocks 1
#define NThreads 10 // rows

__global__ void kernel(int *a, int *b, int *c){
	// These ones will compute elements from [0..2047][0..2047]
	int x = threadIdx.x + blockIdx.x * blockDim.x; // [0..19]
	int y = threadIdx.y + blockIdx.y * blockDim.y; // [0..19]

	// We want to be able to compute as well data between [2047..X]
//	while((x < N) && (y < N)){
//		c[x+y*NThreads] = a[x+y*NThreads] + b[x+y*NThreads]; ;
//		x += blockDim.x * gridDim.x;
//		y += blockDim.x * gridDim.y;
//	}
	if(x<N && y<N){
		c[x+y*NThreads] = a[x+y*NThreads] + b[x+y*NThreads];
	}
}

int main( void ){
	// Variables
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int i;
	int result = 0;
	hipError_t cuda_ret;

	// Reserve memory CPU
	a = (int * ) malloc(N * N * sizeof(int));
	b = (int * ) malloc(N * N * sizeof(int));
	c = (int * ) malloc(N * N * sizeof(int));

	// Reserve memory GPU
	cuda_ret = hipMalloc((void**) &d_a, sizeof(int) * N * N);
	if(cuda_ret != hipSuccess) printf("Unable to allocate device memory");
	cuda_ret = hipMalloc((void**) &d_b, sizeof(int) * N * N);
	if(cuda_ret != hipSuccess) printf("Unable to allocate device memory");
	cuda_ret = hipMalloc((void**) &d_c, sizeof(int) * N * N);
	if(cuda_ret != hipSuccess) printf("Unable to allocate device memory");

	// Init values for the matrix
	//initMatrixes(&a[0][0], &b[0][0]);
	for(i=0; i<N; i++){
		for(int j=0; j<N; j++){
			a[i*N+j] = 1;
			b[i*N+j] = 2;
		}
	}

	for(i=0; i<N; i++){
		for(int j=0; j<N; j++){
			printf("%d", a[i+j*N]+b[i+j*N]);
		}
		printf("\n");
	}

	// Copy values to the GPU
	cuda_ret = hipMemcpy(d_a, a, sizeof(int) * N * N, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) printf("Unable to allocate device memory");
	cuda_ret = hipMemcpy(d_b, b, sizeof(int) * N * N, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) printf("Unable to allocate device memory");

	dim3 threads(NThreads, NThreads);
	// Process data in GPU
	kernel<<<NBlocks,threads>>>(d_a, d_b, d_c);
		
	// Data to Mem CPU
	hipMemcpy(c, d_c, sizeof(int) * N * N, hipMemcpyDeviceToHost);
	
	int j;
	for(i=0; i < N; i++){
		result = 0;
		for(j=0; j<N; j++){
			result += c[i*N+j];
			printf("%d,", c[i*N+j]);
		}
		printf("----> %d", result);
		printf("\n");
	}

//	// Print success
	printf("Result by row: %d\n", result);

	// Free memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
